#include "hip/hip_runtime.h"
/*
	A GPU implementation of Bayesian Personalized Ranking
	Created by Ashley Wang
*/


#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include <math.h>
#include <ctime>
#include <cstdlib>
#include "gputimer.h"


using namespace std;


#define NUM_USERS 2000
#define NUM_ITEMS 5000
#define NUM_SAMPLES 86670
#define RANK 128


struct triple {
	int uid;
	int iid_seen;
	int iid_unseen;
};

struct embedding {
	float vals[RANK];
};


void print_embedding(embedding *embed) {
	printf("{ ");
	for (int i = 0; i < RANK; i++)  { printf("%.3f ", embed->vals[i]); }
		printf("}\n");
}


__inline__ __device__
float dot(const embedding * a, const embedding * b) {
	float val = 0;
	for(int i=0; i<RANK; i++) {
		float a_val = a->vals[i], b_val = b->vals[i];
		val += a_val* b_val;
	}
	return val;
}


__global__ void bpr_update_kernel(triple * user_items, embedding * user_mat, embedding * prod_mat, float alpha, float lambda) {

	__shared__ embedding shared_memory;
	embedding * temp = &shared_memory;

	for (int i = blockIdx.x; i < NUM_SAMPLES; i += gridDim.x) {
		int uid = user_items[i].uid,
			iid_seen = user_items[i].iid_seen, 
			iid_unseen = user_items[i].iid_unseen;

		embedding *user = &user_mat[uid],
			*item_seen = &prod_mat[iid_seen],
			*item_unseen = &prod_mat[iid_unseen];

		float user_val = user->vals[threadIdx.x],
			seen_val = item_seen->vals[threadIdx.x],
			unseen_val = item_unseen->vals[threadIdx.x];

		temp->vals[threadIdx.x] = seen_val - unseen_val;
		__syncthreads();

		float score = dot(user, temp);
		float z = 1.0 / (1.0 + exp(score));

		// if (uid == 29 && iid_seen == 1481 && threadIdx.x == 0) {
		// 	printf("%.3f %.3f\n", score, z);
		// }

		if (z < .5) continue;

		atomicAdd(&user->vals[threadIdx.x], alpha*(z*(seen_val-unseen_val)-lambda*user_val));
		atomicAdd(&item_seen->vals[threadIdx.x], alpha*(z*user_val-lambda*seen_val));
		atomicAdd(&item_unseen->vals[threadIdx.x], alpha*(-z*user_val-lambda*unseen_val));
	}

}

int main(int argc,char **argv) {

	// Setup input data
	ifstream fin("/home/swang3/data/user_item_trim.txt");
	triple user_item_trim[NUM_SAMPLES];
	int i = 0;
	string line;
	while (getline(fin, line)) {
		istringstream iss(line);
		triple temp;
		iss >> temp.uid;
		iss >> temp.iid_seen;
		iss >> temp.iid_unseen;
		user_item_trim[i] = temp;
		i++;
	}
	triple *d_user_item_trim;
	hipMalloc((void **) &d_user_item_trim, NUM_SAMPLES * sizeof(triple));
	hipMemcpy(d_user_item_trim, user_item_trim, NUM_SAMPLES * sizeof(triple), hipMemcpyHostToDevice);

	// Setup user/item embeddings
	embedding user_mat[NUM_USERS], prod_mat[NUM_ITEMS];
	srand(time(NULL));
	for(int i=0; i<NUM_USERS; i++)
		for(int j=0; j<RANK; j++)
			user_mat[i].vals[j] = ((float) rand()/RAND_MAX);

	for(int i=0; i<NUM_ITEMS; i++)
		for(int j=0; j<RANK; j++)
			prod_mat[i].vals[j] = ((float) rand()/RAND_MAX);

	// print_embedding(&user_mat[29]);
	// print_embedding(&prod_mat[1481]);

	embedding *d_user_mat, *d_prod_mat;
	hipMalloc((void **) &d_user_mat, NUM_USERS * sizeof(embedding));
	hipMemcpy(d_user_mat, user_mat, NUM_USERS * sizeof(embedding), hipMemcpyHostToDevice);
	hipMalloc((void **) &d_prod_mat, NUM_ITEMS * sizeof(embedding));
	hipMemcpy(d_prod_mat, prod_mat, NUM_ITEMS * sizeof(embedding), hipMemcpyHostToDevice);

	// Run Matrix Factorization Kernel
	GpuTimer timer;
	timer.Start();
	bpr_update_kernel<<<256, RANK>>>(d_user_item_trim, d_user_mat, d_prod_mat, 0.1, 0.001);
	timer.Stop();

	hipMemcpy(user_mat, d_user_mat, NUM_USERS * sizeof(embedding), hipMemcpyDeviceToHost);
	hipMemcpy(prod_mat, d_prod_mat, NUM_ITEMS * sizeof(embedding), hipMemcpyDeviceToHost);

	hipFree(d_user_item_trim);
	hipFree(d_user_mat);
	hipFree(d_prod_mat);
	printf("Time elapsed = %g ms\n", timer.Elapsed());

	// print_embedding(&user_mat[29]);
	// print_embedding(&prod_mat[1481]);

	return 0;
}